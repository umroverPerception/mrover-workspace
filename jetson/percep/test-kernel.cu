
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__host__ void checkStatus(hipError_t status) {
	if (status != hipSuccess) {
		printf("%s \n", hipGetErrorString(status));

		return;
	}
}

__global__ void fast_add(float* a, float* b, float* res) {
	res[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
	return;
}

int test_kernel(void) {
	float aHost[] = {1, 2, 3, 4, 5};
	float bHost[] = {1, 2, 3, 4, 5};
	float* aDevice, *bDevice, *resDevice, *resHost;

	hipMalloc((void**)&aDevice, sizeof(float)*5);
	hipMalloc((void**)&bDevice, sizeof(float)*5);
	hipMalloc((void**)&resDevice, sizeof(float)*5);
	checkStatus(hipGetLastError());
	resHost = (float*) malloc(sizeof(float)*5);
	hipMemcpy(aDevice, aHost, sizeof(float)*5, hipMemcpyHostToDevice);
	hipMemcpy(bDevice, bHost, sizeof(float)*5, hipMemcpyHostToDevice);
	checkStatus(hipGetLastError());

	fast_add<<<1, 5>>>(aDevice, bDevice, resDevice);
	checkStatus(hipGetLastError());
	hipDeviceSynchronize();
	hipMemcpy(resHost, resDevice, sizeof(float)*5, hipMemcpyDeviceToHost);

	for(int i = 0; i < 5; i++) cout << resHost[i] << endl;

	free(resHost);
	hipFree(resDevice);
	hipFree(aDevice);
	hipFree(bDevice);
	return 0;
}
