#include "hip/hip_runtime.h"
#include "euclidean-cluster.hpp"
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include "common.hpp"
#include <limits>     //for std::numeric_limits<> 

//Helper functions
__device__ float getFloatData(int axis, sl::float4 &val) {
    if(!axis)
        return val.x;
    else if(axis == 1)
        return val.y;
    else
        return val.z;
}
            
__device__ float getData(int axis, int index, sl::float4 *data) {
    return getFloatData(axis, data[index]);    
}

//Hash function that deteremines bin number
__device__ int hashToBin(sl::float4 &data, float* min, float* max, int partitions) {
    int cpx = (data.x-min[0])/(max[0]-min[0])*partitions;
    int cpy = (data.y-min[1])/(max[1]-min[1])*partitions;
    int cpz = (data.z-min[2])/(max[2]-min[2])*partitions;
    return cpx*partitions*partitions+cpy*partitions+cpz;
}


/**
This kernel uses parallel reduction to find the 6 maximum and minimum points
in the point cloud
*/
__global__ void findBoundingBoxKernel(GPU_Cloud_F4 pc, int *minXGlobal, int *maxXGlobal,
                                int *minYGlobal, int *maxYGlobal, int *minZGlobal, int *maxZGlobal){
    //Would it be better to do multiple parallel reductions than one large memory consuming reduction?
    //This method makes 6 copies of the point cloud to find the necessary values 
    const int threads = MAX_THREADS;
    __shared__ int localMinX[threads/2];
    __shared__ int localMaxX[threads/2];
    __shared__ int localMinY[threads/2];
    __shared__ int localMaxY[threads/2];
    __shared__ int localMinZ[threads/2];
    __shared__ int localMaxZ[threads/2];
    __shared__ sl::float4 data[threads];
    __shared__ bool notFull;

    sl::float4 defaultInit(-1.0,-1.0 , -1.0, 0);

    int actualIndex = threadIdx.x + blockIdx.x * blockDim.x;
    

    if(actualIndex < pc.size){ //only write to shared memory if threads about to die
        data[threadIdx.x] = pc.data[actualIndex]; //Write from global memory into shared memory
    }
    else { //Accounts for final block with more threads than points
        notFull = true;
        data[threadIdx.x] = defaultInit;
    }
    __syncthreads();

    int aliveThreads = threads / 2;

    if(!notFull) { //Don't have to worry about checking for going out of bounds
    
        int minX = threadIdx.x, maxX = minX, minY = minX,
        maxY = minX, minZ = minX, maxZ = minZ; //initialize local indices of mins and maxes
        
        //Hard coding first iteration in order to save memory
        if (threadIdx.x < aliveThreads) {
            minX = (data[aliveThreads + threadIdx.x].x < data[minX].x) ? aliveThreads + threadIdx.x : minX;
            maxX = (data[aliveThreads + threadIdx.x].x > data[maxX].x) ? aliveThreads + threadIdx.x : maxX;
            minY = (data[aliveThreads + threadIdx.x].y < data[minY].y) ? aliveThreads + threadIdx.x : minY;
            maxY = (data[aliveThreads + threadIdx.x].y > data[maxY].y) ? aliveThreads + threadIdx.x : maxY;
            minZ = (data[aliveThreads + threadIdx.x].z < data[minZ].z) ? aliveThreads + threadIdx.x : minZ;
            maxZ = (data[aliveThreads + threadIdx.x].z > data[maxZ].z) ? aliveThreads + threadIdx.x : maxZ;
            if (threadIdx.x >= (aliveThreads) / 2) {//Your going to die next iteration, so write to shared
                localMinX[threadIdx.x] = minX;
                localMaxX[threadIdx.x] = maxX;
                localMinY[threadIdx.x] = minY;
                localMaxY[threadIdx.x] = maxY;
                localMinZ[threadIdx.x] = minZ;
                localMaxZ[threadIdx.x] = maxZ;
            }
        }
        __syncthreads();
        aliveThreads /= 2;

        //Utilizes local arrays to keep track of values instead of hardcoded above
        while (aliveThreads > 0) {
            if (threadIdx.x < aliveThreads) {
                minX = (data[localMinX[aliveThreads + threadIdx.x]].x < data[minX].x) ? localMinX[aliveThreads + threadIdx.x] : minX;
                maxX = (data[localMaxX[aliveThreads + threadIdx.x]].x > data[maxX].x) ? localMaxX[aliveThreads + threadIdx.x] : maxX;
                minY = (data[localMinY[aliveThreads + threadIdx.x]].y < data[minY].y) ? localMinY[aliveThreads + threadIdx.x] : minY;
                maxY = (data[localMaxY[aliveThreads + threadIdx.x]].y > data[maxY].y) ? localMaxY[aliveThreads + threadIdx.x] : maxY;
                minZ = (data[localMinZ[aliveThreads + threadIdx.x]].z < data[minZ].z) ? localMinZ[aliveThreads + threadIdx.x] : minZ;
                maxZ = (data[localMaxZ[aliveThreads + threadIdx.x]].z > data[maxZ].z) ? localMaxZ[aliveThreads + threadIdx.x] : maxZ;
                if (threadIdx.x >= (aliveThreads) / 2) {//Your going to die next iteration, so write to shared
                    localMinX[threadIdx.x] = minX;
                    localMaxX[threadIdx.x] = maxX;
                    localMinY[threadIdx.x] = minY;
                    localMaxY[threadIdx.x] = maxY;
                    localMinZ[threadIdx.x] = minZ;
                    localMaxZ[threadIdx.x] = maxZ;
                }
            }
            __syncthreads();
            aliveThreads /= 2;
        }
    }
    
    else{ //Now we have to worry ab having a block that's not full
        int minX = actualIndex < pc.size ? threadIdx.x : -1, maxX = minX, minY = minX,
        maxY = minX, minZ = minX, maxZ = minX; //initialize local indices of mins and maxes checking for those exceeding size
        
        //Hard coding first iteration in order to save memory
        if (threadIdx.x < aliveThreads) {
            
            if(aliveThreads + threadIdx.x + blockDim.x*blockIdx.x < pc.size) { //If points to valid data
                minX = (data[aliveThreads + threadIdx.x].x < data[minX].x) ? aliveThreads + threadIdx.x : minX;
                maxX = (data[aliveThreads + threadIdx.x].x > data[maxX].x) ? aliveThreads + threadIdx.x : maxX;
                minY = (data[aliveThreads + threadIdx.x].y < data[minY].y) ? aliveThreads + threadIdx.x : minY;
                maxY = (data[aliveThreads + threadIdx.x].y > data[maxY].y) ? aliveThreads + threadIdx.x : maxY;
                minZ = (data[aliveThreads + threadIdx.x].z < data[minZ].z) ? aliveThreads + threadIdx.x : minZ;
                maxZ = (data[aliveThreads + threadIdx.x].z > data[maxZ].z) ? aliveThreads + threadIdx.x : maxZ;
            }
            if (threadIdx.x >= (aliveThreads) / 2) {//Your going to die next iteration, so write to shared
                localMinX[threadIdx.x] = minX;
                localMaxX[threadIdx.x] = maxX;
                localMinY[threadIdx.x] = minY;
                localMaxY[threadIdx.x] = maxY;
                localMinZ[threadIdx.x] = minZ;
                localMaxZ[threadIdx.x] = maxZ;
            }
        }
        __syncthreads();
        aliveThreads /= 2;

        //Utilizes local arrays to keep track of values instead of hardcoded above
        while (aliveThreads > 0) {
            if (threadIdx.x < aliveThreads) {
                if(localMinX[aliveThreads + threadIdx.x] >= 0) { //If valid value compare and choose appropriately
                    if(data[localMinX[aliveThreads + threadIdx.x]].x < data[minX].x) minX = localMinX[aliveThreads + threadIdx.x];
                
                    //minX = (data[localMinX[aliveThreads + threadIdx.x]].x < data[minX].x) ? aliveThreads + threadIdx.x : minX;
                    maxX = (data[localMaxX[aliveThreads + threadIdx.x]].x > data[maxX].x) ? localMaxX[aliveThreads + threadIdx.x] : maxX;
                    minY = (data[localMinY[aliveThreads + threadIdx.x]].y < data[minY].y) ? localMinY[aliveThreads + threadIdx.x] : minY;
                    maxY = (data[localMaxY[aliveThreads + threadIdx.x]].y > data[maxY].y) ? localMaxY[aliveThreads + threadIdx.x] : maxY;
                    minZ = (data[localMinZ[aliveThreads + threadIdx.x]].z < data[minZ].z) ? localMinZ[aliveThreads + threadIdx.x] : minZ;
                    maxZ = (data[localMaxZ[aliveThreads + threadIdx.x]].z > data[maxZ].z) ? localMaxZ[aliveThreads + threadIdx.x] : maxZ;
                }
                if (threadIdx.x >= (aliveThreads) / 2) {//Your going to die next iteration, so write to shared
                    localMinX[threadIdx.x] = minX;
                    localMaxX[threadIdx.x] = maxX;
                    localMinY[threadIdx.x] = minY;
                    localMaxY[threadIdx.x] = maxY;
                    localMinZ[threadIdx.x] = minZ;
                    localMaxZ[threadIdx.x] = maxZ;
                }
            }
            __syncthreads();
            aliveThreads /= 2;
        }
    }
    //Write to global memory
    if(threadIdx.x == 0){
        minXGlobal[blockIdx.x] = localMinX[0] + blockDim.x*blockIdx.x;
        maxXGlobal[blockIdx.x] = localMaxX[0] + blockDim.x*blockIdx.x;
        minYGlobal[blockIdx.x] = localMinY[0] + blockDim.x*blockIdx.x;
        maxYGlobal[blockIdx.x] = localMaxY[0] + blockDim.x*blockIdx.x;
        minZGlobal[blockIdx.x] = localMinZ[0] + blockDim.x*blockIdx.x;
        maxZGlobal[blockIdx.x] = localMaxZ[0] + blockDim.x*blockIdx.x;
    }
    return;

}

/**
The final reduction to extrema to find the ultimate extrema from the
provided list. Split into 3 blocks each calculating the max and min 
values for their given axis. Needed to divide it up since float4 = 16bytes
and we have 2048 float4
*/
__global__ void findExtremaKernel (GPU_Cloud_F4 pc, int size, int *minGlobal, int *maxGlobal, 
    float* finalMin, float* finalMax, int axis) {
    
    //Copy from global to shared memory
    const int threads = MAX_THREADS;
    __shared__ int localMin[threads];
    __shared__ int localMax[threads];
    __shared__ sl::float4 localMinData[threads];
    __shared__ sl::float4 localMaxData[threads];
    
    //Copy in all of the local data check for uninitialized values
    //Shouldn't cause warp divergence since the first set of contiguous
    //numbers will enter the else and the second half will enter the if
    
    if(threadIdx.x >= size) {
        localMin[threadIdx.x] = -1;
        localMax[threadIdx.x] = -1;
        localMinData[threadIdx.x] = pc.data[0];
        localMaxData[threadIdx.x] = pc.data[0];
    }
    else {
        localMin[threadIdx.x] = minGlobal[threadIdx.x];
        localMax[threadIdx.x] = maxGlobal[threadIdx.x];
        localMinData[threadIdx.x] = pc.data[localMin[threadIdx.x]];
        localMaxData[threadIdx.x] = pc.data[localMax[threadIdx.x]];
    }
    __syncthreads();

    //Registry memory initializations
    int min = localMin[threadIdx.x];
    int max = localMax[threadIdx.x];
    int aliveThreads = (blockDim.x) / 2;
    sl::float4 minData = localMinData[threadIdx.x];
    sl::float4 maxData = localMaxData[threadIdx.x];

    __syncthreads();

    //Do parallel reduction and modify both values as you go along
    while (aliveThreads > 0) {
        if (threadIdx.x < aliveThreads && localMin[threadIdx.x+aliveThreads] != -1) {
            //Check if value smaller than min
            if(getFloatData(axis, minData) > getFloatData(axis, localMinData[threadIdx.x + aliveThreads])) {
                minData = localMinData[threadIdx.x + aliveThreads];
                min = localMin[threadIdx.x + aliveThreads];
            }
            //Check if value larger than max
            if(getFloatData(axis, maxData) < getFloatData(axis, localMaxData[threadIdx.x + aliveThreads])) {
                maxData = localMaxData[threadIdx.x + aliveThreads];
                max = localMax[threadIdx.x + aliveThreads];
            }

            //Check if thread is going to die next iteration
            if (threadIdx.x >= (aliveThreads) / 2) {
                localMin[threadIdx.x] = min;
                localMax[threadIdx.x] = max;
                localMinData[threadIdx.x] = minData;
                localMaxData[threadIdx.x] = maxData;
            }
        }
        __syncthreads();
        aliveThreads /= 2;
    }

    //If final thread write to global memory
    if(threadIdx.x == 0){
        finalMin[axis] = getFloatData(axis, minData);
        finalMax[axis] = getFloatData(axis, maxData);
        //std::printf("Axis %i min index: %.1f\n", axis, getFloatData(axis, localMinData[threadIdx.x]));
        //std::printf("Axis %i max index: %.1f\n", axis, getFloatData(axis, localMaxData[threadIdx.x]));
        
        //If the last axis calculated readjust so the values make a cube
        if(axis == 2){
            float difX = finalMax[0]-finalMin[0];
            float difY = finalMax[1]-finalMin[1];
            float difZ = finalMax[2]-finalMin[2];
    
            if(difZ >= difY && difZ >= difX) {
                float addY = (difZ-difY)/2+1;
                float addX = (difZ-difX)/2+1;
                finalMax[0] += addX;
                finalMin[0] -= addX;
                finalMax[1] += addY;
                finalMin[1] -= addY; 
                finalMax[2] += 1;
                finalMin[2] -= 1;
            }

            else if(difY >= difX && difY >= difZ) {
                float addZ = (difY-difZ)/2+1;
                float addX = (difY-difX)/2+1;
                finalMax[0] += addX;
                finalMin[0] -= addX;
                finalMax[2] += addZ;
                finalMin[2] -= addZ;
                finalMax[1] += 1;
                finalMin[1] -= 1;
            }

            else {
                float addY = (difX-difY)/2+1;
                float addZ = (difX-difZ)/2+1;
                finalMax[2] += addZ;
                finalMin[2] -= addZ;
                finalMax[1] += addY;
                finalMin[1] -= addY;
                finalMax[0] += 1;
                finalMin[0] -= 1;
            }

        }
    }
      
}

/*
Finds the 6 maximum and minimum points needed to define a bounding box around the 
point cloud. Performs a function 6 times to find each point. The maximum pc size
for this function is 1048576 since it assumes the resulting reduction fits into a block
*/
void EuclideanClusterExtractor::findBoundingBox(GPU_Cloud_F4 &pc){
    const int threads = MAX_THREADS;
    int blocks = ceilDiv(pc.size,threads);
    int *minX; //Stores max and min x,y,z values for each block in global memory
    int *maxX;
    int *minY; 
    int *maxY;
    int *minZ; 
    int *maxZ;

    std::cerr << "Find bound\n";
    checkStatus(hipMalloc(&minX, sizeof(int) * blocks));
    checkStatus(hipMalloc(&maxX, sizeof(int) * blocks));
    checkStatus(hipMalloc(&minY, sizeof(int) * blocks));
    checkStatus(hipMalloc(&maxY, sizeof(int) * blocks));
    checkStatus(hipMalloc(&minZ, sizeof(int) * blocks));
    checkStatus(hipMalloc(&maxZ, sizeof(int) * blocks));

    //Find 6 bounding values for all blocks
    findBoundingBoxKernel<<<blocks,threads>>>(pc, minX, maxX, minY, maxY, minZ, maxZ); 
    checkStatus(hipGetLastError());
    hipDeviceSynchronize();

    //Find X extrema in remaining array
    findExtremaKernel<<<1, threads>>>(pc, blocks, minX, maxX, mins, maxes, 0);
    checkStatus(hipGetLastError());
    hipDeviceSynchronize();

    //Find Y extrema in remaining array
    findExtremaKernel<<<1, threads>>>(pc, blocks, minY, maxY, mins, maxes, 1);
    checkStatus(hipGetLastError());
    hipDeviceSynchronize();

    //Find Z extrema
    findExtremaKernel<<<1, threads>>>(pc, blocks, minZ, maxZ, mins, maxes, 2);
    checkStatus(hipGetLastError());
    hipDeviceSynchronize();

    //Should print out 2,9,0,7,1,6

    //Free memory
    hipFree(minX);
    hipFree(maxX);
    hipFree(minY);
    hipFree(maxY);
    hipFree(minZ);
    hipFree(maxZ);
    std::cerr <<"Find bound complete\n";

}
__global__ void zeroBinsKernel(int* binCount, int partitions) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx < partitions*partitions*partitions){
        binCount[idx] = 0;
    }
}

/*
This kernel will use a hash function to determine which bin the point hashes into
and will then atomically count the number of points to be added to the bin. 
THERE IS DEFINITELY A BETTER WAY TO DO THIS STEP
*/

__global__ void buildBinsKernel(GPU_Cloud_F4 pc, int* binCount, int partitions, 
                                        float* min, float* max, int** bins, int* memo) {
    
    int ptIdx = threadIdx.x + blockDim.x * blockIdx.x;
    if(ptIdx >= pc.size) return;

    //Copy Global to registry memory
    sl::float4 data = pc.data[ptIdx];

    int binNum = hashToBin(data, min, max, partitions);

    //Find total number of elements in each bin
    int place = atomicAdd(&binCount[binNum],1);
   
    //Make intermediary step to write to global memory. Could avoid this by syncing
    //all blocks
    memo[3*ptIdx] = ptIdx;
    memo[3*ptIdx+1] = binNum;
    memo[3*ptIdx+2] = place;
}

__global__ void mallocBinsKernel(int partitions, int** bins, int* binCount) {
    int ptIdx = threadIdx.x + blockDim.x * blockIdx.x;

    //Dynamically allocate memory for bins in kernel. Memory must be freed
    //in a different Kernel. It cannot be freed with hipFree()
    //By definition of the hash function there will be partitions^3 bins 
    if(ptIdx < partitions*partitions*partitions) {
        bins[ptIdx] = (int*)malloc(sizeof(int)*(binCount[ptIdx]));
    }
}

__global__ void assignBinsKernel(int size, int** bins, int* memo) {
    int ptIdx = threadIdx.x + blockDim.x * blockIdx.x;
    if(ptIdx >= size) return;

    //Memory now exists, so write index to global memory
    bins[memo[3*ptIdx+1]][memo[3*ptIdx+2]] = memo[3*ptIdx];

    //printf("(%i, %i, %i), ", memo[3*ptIdx+1], memo[3*ptIdx+2], bins[memo[3*ptIdx+1]][memo[3*ptIdx+2]]);
}

__global__ void freeBinsKernel(int* binCount, int** bins, int partitions){
    
    int ptIdx = threadIdx.x + blockDim.x * blockIdx.x;

    //If valid bin
    if(ptIdx < partitions*partitions*partitions){
        int* ptr = bins[ptIdx];
        //If memory was allocated
        if(ptr != NULL)
            free(ptr);
    }
}


/*
This function builds the bins that are needed to prevent an O(n^2) search time
for nearest neighbors. Uses min and max values to construct a cube that can be 
divided up into a specified number of partitions on each axis. 
*/
void EuclideanClusterExtractor::buildBins(GPU_Cloud_F4 &pc) {
    int threads = MAX_THREADS;
    int blocks = ceilDiv(pc.size, threads);
    int* memo;
    
    //Allocate memory
    checkStatus(hipMalloc(&bins, sizeof(int*) * partitions*partitions*partitions));
    checkStatus(hipMalloc(&binCount, sizeof(int) * partitions*partitions*partitions));
    checkStatus(hipMalloc(&memo, sizeof(int) * 3 * pc.size));
    
    //Zero Bins
    zeroBinsKernel<<<ceilDiv(partitions*partitions*partitions, threads), threads>>>(binCount, partitions);
    checkStatus(hipGetLastError());
    hipDeviceSynchronize();

    //Construct the bins to be used
    buildBinsKernel<<<blocks, threads>>>(pc, binCount, partitions, mins, maxes, bins, memo);
    checkStatus(hipGetLastError());
    hipDeviceSynchronize();

    //Allocates appropriate memory for bins
    //Used because couldn't figure out how to sync blocks
    mallocBinsKernel<<<ceilDiv(partitions*partitions*partitions, threads), threads>>>(partitions, bins, binCount);
    checkStatus(hipGetLastError());
    hipDeviceSynchronize();

    //Assign values to the created bin structure
    //Used because couldn't figure out how to sync blocks
    assignBinsKernel<<<blocks, threads>>>(pc.size, bins, memo);
    checkStatus(hipGetLastError());
    hipDeviceSynchronize();

    //Should print something like (5, 0, 0), (6, 0, 1), (3, 0, 2), (7, 0, 3), (0, 0, 4), (3, 1, 5), (1, 0, 6), (2, 0, 7), (6, 1, 8), (1, 1, 9)
    //Don't worry if middle number differs

    //Free memory
    checkStatus(hipFree(memo));
}

/*
This function frees dynamically allocated memory in buildBins function
*/
void EuclideanClusterExtractor::freeBins() {
    int threads = MAX_THREADS;
    int blocks = ceilDiv(partitions*partitions*partitions, threads);
    
    freeBinsKernel<<<blocks,threads>>>(binCount, bins, partitions);
    checkStatus(hipGetLastError());
    hipDeviceSynchronize();

    checkStatus(hipFree(binCount));
    checkStatus(hipFree(bins));
}

__device__ void findEdgePtsOfRadius (sl::float4 &startBinPt, sl::float4 &xBoundPt, sl::float4 &yBoundPt, 
                                    sl::float4 &zBoundPt, int tolerance, float* mins, float* maxes) {
    //Start Bin checking for going out of bounds
    startBinPt.z = (mins[2] < startBinPt.z-tolerance) ? startBinPt.z-tolerance : mins[2]+1;
    startBinPt.y = (mins[1] < startBinPt.y-tolerance) ? startBinPt.y-tolerance : mins[1]+1;
    startBinPt.x = (mins[0] < startBinPt.x-tolerance) ? startBinPt.x-tolerance : mins[0]+1; 
    
    //X Edge checking for going out of bounds
    xBoundPt.z = (mins[2] < xBoundPt.z-tolerance) ? xBoundPt.z-tolerance : mins[2]+1;
    xBoundPt.y = (mins[1] < xBoundPt.y-tolerance) ? xBoundPt.y-tolerance : mins[1]+1;
    xBoundPt.x = (maxes[0] > xBoundPt.x+tolerance) ? xBoundPt.x+tolerance : maxes[0]-1; 
    
    //Y Edge checking for going out of bounds
    yBoundPt.z = (mins[2] < yBoundPt.z-tolerance) ? yBoundPt.z-tolerance : mins[2]+1;
    yBoundPt.y = (maxes[1] > yBoundPt.y+tolerance) ? yBoundPt.y+tolerance : maxes[1]-1;
    yBoundPt.x = (mins[0] < yBoundPt.x-tolerance) ? yBoundPt.x-tolerance : mins[0]+1;                                             
    
    //Z Edge checking for going out of bounds
    zBoundPt.z = (maxes[2] > zBoundPt.z+tolerance) ? zBoundPt.z+tolerance : maxes[2]-1;
    zBoundPt.y = (mins[1] < zBoundPt.y-tolerance) ? zBoundPt.y-tolerance : mins[1]+1;
    zBoundPt.x = (mins[0] < zBoundPt.x-tolerance) ? zBoundPt.x-tolerance : mins[0]+1;                                             
}

/*
This kernel determines the structure of the graph but does not build it
In theory, there is a memory-compute trade off to be made here. This kernel
is not strictly necessary if we allow an upper bound of memory so that each 
point can have the entire dataset amount of neighbors. Perhaps we can 
explore this allocation method instead.
*/
//b: enough, t: each point
__global__ void determineGraphStructureKernel(GPU_Cloud_F4 pc, float tolerance, int* listStart, int** bins, int* binCount,
                                            float* mins, float* maxes, int partitions) {
    int ptIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(ptIdx >= pc.size) return;

    sl::float3 pt = pc.data[ptIdx];
    int neighborCount = 0;

    //Initialize edge vars
    sl::float4 startBinPt = pc.data[ptIdx], xBoundPt = startBinPt, 
    yBoundPt = startBinPt, zBoundPt = startBinPt;

    findEdgePtsOfRadius(startBinPt, xBoundPt, yBoundPt, zBoundPt, tolerance, mins, maxes);

    //Find Edge Bins
    int xStartBin = hashToBin(startBinPt, mins, maxes, partitions);
    int xBoundBin = hashToBin(xBoundPt, mins, maxes, partitions);
    int yBoundBin = hashToBin(yBoundPt, mins, maxes, partitions);
    int zBoundBin = hashToBin(zBoundPt, mins, maxes, partitions);

    int yStartBin = xStartBin, zStartBin = xStartBin;

    const int totalBins = (zBoundBin-xStartBin+1) * ((yBoundBin-xStartBin)/partitions+1) *
                    ((xBoundBin-xStartBin)/(partitions*partitions)+1);

    int* binsToSearch = (int*)malloc(sizeof(int)*totalBins);

    /*
    The bin one away from currBin in Z direction is binNum +-1
    The bin one away from currBin in Y direction is binNum +-partitions
    The bin one away from currBin in X direction is binNum +-partitions^2
    */
    //Start at lower left, iterate front to back, bottom to top, left to right 
    //Iterate left to right
    int binAdded = 0;
    int zDif = zBoundBin-zStartBin;
    for(int i = xStartBin; i <= xBoundBin; i += partitions*partitions) {
        //Iterate bottom to top
        for(int j = yStartBin; j <= yBoundBin; j += partitions) {
            //Iterate front to back
            for(int k = zStartBin; k <= zBoundBin; ++k){
                binsToSearch[binAdded] = k;
                binAdded++;
            }
            zBoundBin += partitions; //Shift zBoundBin up
            zStartBin += partitions; //Shift ztartBin up
        }
        yBoundBin += (partitions*partitions); //Shift yBoundBin right
        yStartBin += (partitions*partitions); //Shift yStartBin right
        zStartBin = yStartBin;
        zBoundBin = zStartBin+zDif;
    }

    //Iterate through points in bins to search and check if they are within the radius of the point
    for(size_t i = 0; i < totalBins; ++i){
        for(int j = 0; j < binCount[binsToSearch[i]]; ++j){
            sl::float3 dvec = (pt - sl::float3(pc.data[bins[binsToSearch[i]][j]]));
            
            //this is a neighbor
            if( dvec.norm() < tolerance && bins[binsToSearch[i]][j] != ptIdx) {
                neighborCount++;
            }
        }
    }
    listStart[ptIdx] = neighborCount;
    free(binsToSearch);
    //we must do an exclusive scan using thrust after this kernel
    //printf("%d: %d \n",ptIdx, neighborCount );
}


/* This kernel builds the graph 
Fairly standard adjacency list structure. 
*/
__global__ void buildGraphKernel(GPU_Cloud_F4 pc, float tolerance, int* neighborLists, int* listStart, int* labels, bool* f1, bool* f2, 
                            int** bins, int* binCount, float* mins, float* maxes, int partitions) {
    int ptIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(ptIdx >= pc.size) return;

    sl::float3 pt = pc.data[ptIdx];
    int neighborCount = 0;
    //get the adjacency list for this point
    int* list = neighborLists + listStart[ptIdx];
    
     //Initialize edge vars
    sl::float4 startBinPt = pc.data[ptIdx], xBoundPt = startBinPt, 
    yBoundPt = startBinPt, zBoundPt = startBinPt;

    findEdgePtsOfRadius(startBinPt, xBoundPt, yBoundPt, zBoundPt, tolerance, mins, maxes);

    //Find Edge Bins
    int xStartBin = hashToBin(startBinPt, mins, maxes, partitions);
    int xBoundBin = hashToBin(xBoundPt, mins, maxes, partitions);
    int yBoundBin = hashToBin(yBoundPt, mins, maxes, partitions);
    int zBoundBin = hashToBin(zBoundPt, mins, maxes, partitions);

    int yStartBin = xStartBin, zStartBin = xStartBin;

    const int totalBins = (zBoundBin-xStartBin+1) * ((yBoundBin-xStartBin)/partitions+1) *
                    ((xBoundBin-xStartBin)/(partitions*partitions)+1);

    int* binsToSearch = (int*)malloc(sizeof(int)*totalBins);

    /*
    The bin one away from currBin in Z direction is binNum +-1
    The bin one away from currBin in Y direction is binNum +-partitions
    The bin one away from currBin in X direction is binNum +-partitions^2
    */
    //Start at lower left, iterate front to back, bottom to top, left to right 
    //Iterate left to right
    int binAdded = 0;
    int zDif = zBoundBin-zStartBin;
    for(int i = xStartBin; i <= xBoundBin; i += partitions*partitions) {
        //Iterate bottom to top
        for(int j = yStartBin; j <= yBoundBin; j += partitions) {
            //Iterate front to back
            for(int k = zStartBin; k <= zBoundBin; ++k){
                binsToSearch[binAdded] = k;
                binAdded++;
            }
            zBoundBin += partitions; //Shift zBoundBin up
            zStartBin += partitions; //Shift ztartBin up
        }
        yBoundBin += (partitions*partitions); //Shift yBoundBin right
        yStartBin += (partitions*partitions); //Shift yStartBin right
        zStartBin = yStartBin;
        zBoundBin = zStartBin+zDif;
    }

    //Iterate through points in bins to search and check if they are within the radius of the point
    for(size_t i = 0; i < totalBins; ++i){
        for(int j = 0; j < binCount[binsToSearch[i]]; ++j){
            sl::float3 dvec = (pt - sl::float3(pc.data[bins[binsToSearch[i]][j]]));
            
            //this is a neighbor
            if( dvec.norm() < tolerance && bins[binsToSearch[i]][j] != ptIdx) {
                list[neighborCount] = bins[binsToSearch[i]][j]; 
                neighborCount++;
            }
        }
    }

    free(binsToSearch);
    //we must do an exclusive scan using thrust after this kernel
    //printf("%d: %d \n",ptIdx, neighborCount );

    labels[ptIdx] = ptIdx;
    f1[ptIdx] = true;
    f2[ptIdx] = false;
}

__global__ void determineGraphStructureKernelN2(GPU_Cloud_F4 pc, float tolerance, int* listStart) {
    int ptIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(ptIdx >= pc.size) return;

    sl::float3 pt = pc.data[ptIdx];
    int neighborCount = 0;
    
    //horrible slow way of doing this that is TEMPORARY --> please switch to radix sorted bins
    for(int i = 0; i < pc.size; i++) {
        sl::float3 dvec = (pt - sl::float3(pc.data[i]));
        //this is a neighbor
        if( dvec.norm() < tolerance && i != ptIdx) {
            neighborCount++;
        }
    }
    listStart[ptIdx] = neighborCount;

    //we must do an exclusive scan using thrust after this kernel
    //printf("%d: %d \n",ptIdx, neighborCount );
}


/* This kernel builds the graph 
Fairly standard adjacency list structure. 
*/
__global__ void buildGraphKernelN2(GPU_Cloud_F4 pc, float tolerance, int* neighborLists, int* listStart, int* labels, bool* f1, bool* f2) {
    int ptIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(ptIdx >= pc.size) return;

    sl::float3 pt = pc.data[ptIdx];
    int neighborCount = 0;
    //get the adjacency list for this point
    int* list = neighborLists + listStart[ptIdx];
    
    //horrible slow way of doing this that is TEMPORARY --> please switch to radix sorted bins
    for(int i = 0; i < pc.size; i++) {

        sl::float3 dvec = (pt - sl::float3(pc.data[i]));
        //this is a neighbor
        if( dvec.norm() < tolerance && i != ptIdx) {
            list[neighborCount] = i;
            neighborCount++;
        }
    }
    
    labels[ptIdx] = ptIdx;
    f1[ptIdx] = true;
    f2[ptIdx] = false;
}

/*
this kernel propogates labels, it must be called in a loop until its flag "m" is false, indicating
no more changes are pending. 
*/
//each thread is a point 
__global__ void propogateLabels(GPU_Cloud_F4 pc, int* neighborLists, int* listStart, int* labels, bool* f1, bool* f2, bool* m) {
    int ptIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(ptIdx >= pc.size) return;

    if(ptIdx == -1){
        for(int i = 0; i < 10; i++){
            printf("Pt %i: ", i);
            for(int j = listStart[i]; j < listStart[i+1]; ++j){
                printf("%i, ", neighborLists[j]);
            }
            printf("\n");    
        }
        
    }
    //debug lines
   // if(threadIdx.x == 0) *m = false;
   // __syncthreads();
   // printf("pt idx: %d, label: %d, flag: %d frontier one: %d frontier two: %d \n", ptIdx, labels[ptIdx], (*m) ? 1 : 0, f1[ptIdx] ? 1 : 0, f2[ptIdx] ? 1 : 0);

    bool mod = false;
    //TODO, load the NEIGHBOR list to shared memory 
    if(f1[ptIdx]) {
        //printf("active frontier %d \n", ptIdx);

        int* list = neighborLists + listStart[ptIdx];
        int listLen = listStart[ptIdx+1] - listStart[ptIdx];
        f1[ptIdx] = false;
        int myLabel = labels[ptIdx];

        //printf("[len] pt idx: %d, list-len: %d \n", ptIdx, listLen);

        for(int i = 0; i < listLen; i++) {
            int otherLabel = labels[list[i]];
            if(myLabel < otherLabel) { //are these reads actually safe?
                //printf("-- updating other: %d to be %d \n", otherLabel, myLabel);

                atomicMin(&labels[list[i]], myLabel);
                f2[list[i]] = true;
                *m = true;
            } else if(myLabel > otherLabel) {
                myLabel = otherLabel;
                mod = true;
            }
        }

        if(mod) {
            atomicMin(&labels[ptIdx], myLabel);
            f2[ptIdx] = true;
            *m = true;
        }
    } 

    /*
    __syncthreads();
    if(threadIdx.x == 0) {
    if(*m) printf("still going \n");
    else printf("done \n");
    }*/
}

__device__ __forceinline__ float atomicMinFloat (float * addr, float value) {
    float old;
    old = (value >= 0) ? __int_as_float(atomicMin((int *)addr, __float_as_int(value))) :
         __uint_as_float(atomicMax((unsigned int *)addr, __float_as_uint(value)));

    return old;
}


__device__ __forceinline__ float atomicMaxFloat (float * addr, float value) {
    float old;
    old = (value >= 0) ? __int_as_float(atomicMax((int *)addr, __float_as_int(value))) :
         __uint_as_float(atomicMin((unsigned int *)addr, __float_as_uint(value)));

    return old;
}

//this debug kernel colors points based on their label
__global__ void colorClusters(GPU_Cloud_F4 pc, int* labels, int* keys, int* values, int minCloudSize, int numClusters, float* minX, float* maxX, float* minY, float* maxY, float* minZ, float* maxZ) {
    int ptIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(ptIdx >= pc.size) return;

    //DEBUG STEP REMOVE
    //pc.data[ptIdx].w = 9.18340948595e-41;
    //return;

    int i = 0;
    while(true) {
        if(labels[ptIdx] == keys[i]) {
            if(values[i] < minCloudSize) {
                pc.data[ptIdx].w = VIEWER_BGR_COLOR;
                return;
            }
            else break;
        }
        i++;
    }
    
    //float red = 3.57331108403e-43;
    //float green = 9.14767637511e-41;
    //float blue = 2.34180515203e-38;
    //float magenta = 2.34184088514e-38; 
    float yellow = 9.18340948595e-41;
    
    pc.data[ptIdx].w = yellow+0.0000000000000001*labels[ptIdx]*4;
    
    //X
    atomicMinFloat(&minX[i], pc.data[ptIdx].x);
    atomicMaxFloat(&maxX[i], pc.data[ptIdx].x);

    //Y
    atomicMinFloat(&minY[i], pc.data[ptIdx].y);
    atomicMaxFloat(&maxY[i], pc.data[ptIdx].y);

    //Z
    atomicMinFloat(&minZ[i], pc.data[ptIdx].z);
    atomicMaxFloat(&maxZ[i], pc.data[ptIdx].z);
}

//this is practically serial, can we just color using OpenGL functions
__global__ void colorExtrema(GPU_Cloud_F4 pc, int* values, int minSize, int* labels, int numClustersOrig, int* validClustersCount, float* minX, float* maxX,  float* minY, float* maxY, float* minZ, float* maxZ) {
    int clusterIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(clusterIdx >= numClustersOrig) return;

    int place = 0;
    if(values[clusterIdx] > minSize) place = atomicAdd(validClustersCount, 1);
    else return;

  
    pc.data[place*2] = sl::float4(minX[clusterIdx], (minY[clusterIdx] + maxY[clusterIdx])/2, minZ[clusterIdx], 0.0);
    pc.data[place*2+1] = sl::float4(maxX[clusterIdx], (minY[clusterIdx] + maxY[clusterIdx])/2, minZ[clusterIdx], 0.0);
        
    //serailze the extrema into a float4 vector using the "place"
}

__global__ void colorClustersNew(GPU_Cloud_F4 pc, int* labels, int* keys, int numClusters) {
    int ptIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(ptIdx >= pc.size) return;

    float yellow = 9.18340948595e-41;

    for(int i = 0; i < numClusters; i++) {
        if(labels[ptIdx] == keys[i]) {
            pc.data[ptIdx].w = yellow+0.0000000000000001*labels[ptIdx]*4;
            return;
        }
    }

    pc.data[ptIdx].w = VIEWER_BGR_COLOR;

}

    
class is_smaller_than_min {
public: 
    is_smaller_than_min(int min) : min(min) {}
    __device__ __host__ bool operator()(const int size) {
        return size < min;
    }
private:
    int min;
};

EuclideanClusterExtractor::EuclideanClusterExtractor(float tolerance, int minSize, float maxSize, size_t cloudArea, int partitions) 
: tolerance{tolerance}, minSize{minSize}, maxSize{maxSize}, partitions{partitions} {

    hipMalloc(&listStart, sizeof(int)*(cloudArea+1));
    hipMalloc(&labels, sizeof(int)*cloudArea);
    hipMalloc(&f1, sizeof(bool)*cloudArea);
    hipMalloc(&f2, sizeof(bool)*cloudArea);
    hipMalloc(&stillGoing, sizeof(bool));

    //Nearest Neighbor Bins
    checkStatus(hipMalloc(&mins, sizeof(int) * 3));
    checkStatus(hipMalloc(&maxes, sizeof(int) * 3));

   // colorClusters<<<ceilDiv(pc.size, MAX_THREADS), MAX_THREADS>>>(pc, nullptr);
}
EuclideanClusterExtractor::EuclideanClusterExtractor() {

};

//perhaps use dynamic parallelism 
EuclideanClusterExtractor::ObsReturn EuclideanClusterExtractor::extractClusters(GPU_Cloud_F4 pc) {
    ObsReturn empty;
    empty.size = 0;
    if(pc.size == 0) return empty;
    //set frontier arrays appropriately [done in build graph]
    //checkStatus(hipMemsetAsync(f1, 1, sizeof(pc.size)));
    //checkStatus(hipMemsetAsync(f2, 0, sizeof(pc.size)));
    std::cerr <<"Determining Graph Structure\n";
    //determineGraphStructureKernel<<<ceilDiv(pc.size, MAX_THREADS), MAX_THREADS>>>(pc, tolerance, listStart, bins, binCount, mins, maxes, partitions);
    determineGraphStructureKernelN2<<<ceilDiv(pc.size, MAX_THREADS), MAX_THREADS>>>(pc, tolerance, listStart);
    std::cerr <<"Structure Determined\n";
    thrust::exclusive_scan(thrust::device, listStart, listStart+pc.size+1, listStart, 0);
    checkStatus(hipGetLastError());
    checkStatus(hipDeviceSynchronize());
    int totalAdjanecyListsSize;
    /*//debugint* temp = (int*) malloc(sizeof(int)*(pc.size+1));
    checkStatus(hipMemcpy(temp, listStart, sizeof(int)*(pc.size+1), hipMemcpyDeviceToHost));
    for(int i = 0; i < pc.size+1; i++) std::cout << "ex scan: " << temp[i] << std::endl; */
    checkStatus(hipMemcpy(&totalAdjanecyListsSize, &listStart[pc.size], sizeof(int), hipMemcpyDeviceToHost));
    //std::cout << "total adj size: " << totalAdjanecyListsSize << std::endl;
    std::cerr<<"Building graph kernel\n";
    hipMalloc(&neighborLists, sizeof(int)*totalAdjanecyListsSize);
    //buildGraphKernel<<<ceilDiv(pc.size, MAX_THREADS), MAX_THREADS>>>(pc, tolerance, neighborLists, listStart, labels, f1, f2,
      //                                  bins, binCount, mins, maxes, partitions);
    buildGraphKernelN2<<<ceilDiv(pc.size, MAX_THREADS), MAX_THREADS>>>(pc, tolerance, neighborLists, listStart, labels, f1, f2);
    std::cerr<<"Graph kernel built\n";
    checkStatus(hipGetLastError());
    checkStatus(hipDeviceSynchronize());
    

    
    bool stillGoingCPU = true;    
    while(stillGoingCPU) {
        //one iteration of label propogation
        stillGoingCPU = false;
        hipMemcpy(stillGoing, &stillGoingCPU, sizeof(bool), hipMemcpyHostToDevice);
        propogateLabels<<<ceilDiv(pc.size, MAX_THREADS), MAX_THREADS>>>(pc, neighborLists, listStart, labels, f1, f2, stillGoing);

        //swap the frontiers
        bool* t = f1;
        f1 = f2;
        f2 = t;

        //get flag to see if we are done
        hipMemcpy(&stillGoingCPU, stillGoing, sizeof(bool), hipMemcpyDeviceToHost);
    }

    //Build useful data structures mapping points to clusters and clusters to number of points
    //Let C be the number of clusters, and N the number of points in the cloud
    //After we preform the operations in this block, the contents of the vectors are as follows:
    thrust::device_vector<int> labelsSorted(pc.size); //Point labels sorted by cluster. Len(N). 
    thrust::device_vector<int> count(pc.size, 1); //buffer of all 1s. Len(N)
    thrust::device_vector<int> keys(pc.size); //Each clusters unique ID in ascending order Len(C)
    thrust::device_vector<int> values(pc.size); //The number of points in each cluster in ascending order by ID. Len(C)

    thrust::copy(thrust::device, labels, labels+pc.size, labelsSorted.begin()); //first make the labels sorted contain the labels in order of points
    thrust::sort(thrust::device, labelsSorted.begin(), labelsSorted.end()); //now sort the labels by their label idx, 
    auto pair = thrust::reduce_by_key(thrust::device, labelsSorted.begin(), labelsSorted.end(), count.begin(), keys.begin(), values.begin()); //remove duplicate labels and determine the number of points belonging to each label    
   
    //Determine how many clusters there actually are
    
    int numClustersOrig = thrust::distance(keys.begin(), pair.first);
    std::cout << "CLUSTERS ORIG: " << numClustersOrig << std::endl; 

    float *minX, *maxX, *minY, *maxY, *minZ, *maxZ; 
    hipMalloc(&minX, sizeof(float)*numClustersOrig);
    hipMalloc(&maxX, sizeof(float)*numClustersOrig);
    hipMalloc(&minY, sizeof(float)*numClustersOrig);
    hipMalloc(&maxY, sizeof(float)*numClustersOrig);
    hipMalloc(&minZ, sizeof(float)*numClustersOrig);
    hipMalloc(&maxZ, sizeof(float)*numClustersOrig);
    thrust::fill(thrust::device, minX, minX + numClustersOrig, std::numeric_limits<float>::max());
    thrust::fill(thrust::device, maxX, maxX + numClustersOrig, -std::numeric_limits<float>::max());
    thrust::fill(thrust::device, minY, minY + numClustersOrig, std::numeric_limits<float>::max());
    thrust::fill(thrust::device, maxY, maxY + numClustersOrig, -std::numeric_limits<float>::max());
    thrust::fill(thrust::device, minZ, minZ + numClustersOrig, std::numeric_limits<float>::max());
    thrust::fill(thrust::device, maxZ, maxZ + numClustersOrig, -std::numeric_limits<float>::max());

    /*
    //Now get a list of cluster ID keys that are bigger than the min size by removing those that are less than the min size
    is_smaller_than_min pred(minSize);
    auto keyEnd = thrust::remove_if(thrust::device, keys.begin(), keys.end(), values.begin(), pred);
    thrust::remove_if(thrust::device, values.begin(), values.end(), pred);

    int numClusters = keyEnd - keys.begin();
    keys.resize(numClusters);
    values.resize(numClusters);
    std::cout << "CLUSTERS NEW: " << numClusters << std::endl; */

    //find interest points
    //exculsive scan on values to give the indicies of each new cluster start in the points array 
    //for each on the array returned by the exclusive scan, going from the prev element to the cur,
    //first determine if the labels for that range are contained within the clusterIDs [keys] vector (binary search),
    //if so, then find extrema, otherwise move on

    //Call a kernel to color the clusters for debug reasons
    int* gpuKeys = thrust::raw_pointer_cast( keys.data() );
    int* gpuVals = thrust::raw_pointer_cast( values.data() );
    colorClusters<<<ceilDiv(pc.size, MAX_THREADS), MAX_THREADS>>>(pc, labels, gpuKeys, gpuVals, minSize, numClustersOrig, minX, maxX, minY, maxY, minZ, maxZ);

    int * validClustersCount;
    hipMalloc(&validClustersCount, sizeof(int));
    hipMemset(validClustersCount, 0, sizeof(int));
    //colorExtrema<<<ceilDiv(numClustersOrig, MAX_THREADS), MAX_THREADS >>>(pc, gpuVals, minSize, labels, numClustersOrig, validClustersCount, minX, maxX, minY, maxY, minZ, maxZ);

    float *minXCPU, *maxXCPU, *minYCPU, *maxYCPU, *minZCPU, *maxZCPU; 
    minXCPU = (float*) malloc(sizeof(float)*numClustersOrig);
    maxXCPU = (float*) malloc(sizeof(float)*numClustersOrig);
    minYCPU = (float*) malloc(sizeof(float)*numClustersOrig);
    maxYCPU = (float*) malloc(sizeof(float)*numClustersOrig);
    minZCPU = (float*) malloc(sizeof(float)*numClustersOrig);
    maxZCPU = (float*) malloc(sizeof(float)*numClustersOrig);
    hipMemcpy(minXCPU, minX, sizeof(float)*numClustersOrig, hipMemcpyDeviceToHost);
    hipMemcpy(maxXCPU, maxX, sizeof(float)*numClustersOrig, hipMemcpyDeviceToHost);
    hipMemcpy(minYCPU, minY, sizeof(float)*numClustersOrig, hipMemcpyDeviceToHost);
    hipMemcpy(maxYCPU, maxY, sizeof(float)*numClustersOrig, hipMemcpyDeviceToHost);
    hipMemcpy(minZCPU, minZ, sizeof(float)*numClustersOrig, hipMemcpyDeviceToHost);
    hipMemcpy(maxZCPU, maxZ, sizeof(float)*numClustersOrig, hipMemcpyDeviceToHost);

    int* leftBearing;
    int* rightBearing;
    int* leftCPU;
    int* rightCPU; 

    leftCPU = (int*) malloc(sizeof(int));
    rightCPU = (int*) malloc(sizeof(int));

    hipMalloc(&leftBearing, sizeof(float));
    hipMalloc(&rightBearing, sizeof(float));
    
    //Laucnh kernels to find clear paths using mins and max cluster arrasy
    findClearPathKernel<<<1, MAX_THREADS>>>(minX, maxX, minZ, maxZ, numClustersOrig, leftBearing, rightBearing);
    checkStatus(hipGetLastError());
    hipDeviceSynchronize();

    findAngleOffCenterKernel<<<1, MAX_THREADS>>>(minX, maxX, minZ, maxZ, numClustersOrig, leftBearing, 0);
    checkStatus(hipGetLastError());
    hipDeviceSynchronize();

    findAngleOffCenterKernel<<<1, MAX_THREADS>>>(minX, maxX, minZ, maxZ, numClustersOrig, rightBearing, 1);    
    checkStatus(hipGetLastError());
    hipDeviceSynchronize();
    
    //Copy bearings to CPU and display the bearings
    hipMemcpy(leftCPU, leftBearing, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(rightCPU, rightBearing, sizeof(int), hipMemcpyDeviceToHost);
    
    //Set CPU bearings for viewer use
    bearingRight = *rightCPU;
    bearingLeft = *leftCPU;
    
    hipFree(leftBearing);
    hipFree(rightBearing);
    free(leftCPU);
    free(rightCPU);
    
    checkStatus(hipDeviceSynchronize()); //not needed?
    hipFree(neighborLists);
    hipFree(minX);
    hipFree(maxX);
    hipFree(minY);
    hipFree(maxY);
    hipFree(minZ);
    hipFree(maxZ);

    int validClustersCPU;
    hipMemcpy(&validClustersCPU, validClustersCount, sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "valid cluster size: " << validClustersCPU << std::endl;

    ObsReturn obsReturn;
    obsReturn.size = numClustersOrig;
    obsReturn.minX = minXCPU;
    obsReturn.maxX = maxXCPU;
    obsReturn.minY = minYCPU;
    obsReturn.maxY = maxYCPU;
    obsReturn.minZ = minZCPU;
    obsReturn.maxZ = maxZCPU;
    return obsReturn;
}

EuclideanClusterExtractor::~EuclideanClusterExtractor() {
    checkStatus(hipFree(mins));
    checkStatus(hipFree(maxes));
}
